#include "hip/hip_runtime.h"
#include "bits/stdc++.h"
#include "Matrix.cu"

using namespace std;

__global__ void debugCOO(MatrixCOO * matrix) {
	for(int i = 0; i < matrix->nnz; i++) {
		printf("%d %d %lf\n", matrix->i[i], matrix->j[i], matrix->val[i]);
	}
}

__global__ void debugCSR(MatrixCSR * matrix) {
	for(int i = 0; i < matrix->rows; i++) {
		for(int j = 0; j < matrix->cols; j++) {
			printf("%lf ", getElementMatrixCSR(matrix, i, j));
		}
		printf("\n");
	}
}

__global__ void debugCSC(MatrixCSC * matrix) {
	for(int i = 0; i < matrix->rows; i++) {
		for(int j = 0; j < matrix->cols; j++) {
			printf("%lf ", getElementMatrixCSC(matrix, i, j));
		}
		printf("\n");
	}
}

__global__ void debugmuij(MatrixCSR * matrix, double * Si) {
	for(int i = 0; i < matrix->rows; i++) {
		for(int j = 0; j < matrix->cols; j++) {
			printf("%d %d %lf\n", i, j, muij(i, j, matrix, Si));
		}
	}
}

int main() {

	string filename = "../../matrices/SmallTestMatrix.mtx";

	auto tempCSRCPU = readMatrixCPUMemoryCSR(filename);
	auto tempCSCCPU = readMatrixCPUMemoryCSC(filename);

	auto tempCSR = readMatrixGPUMemoryCSR(filename);
	debugCSR <<<1,1>>> (tempCSR);
	hipDeviceSynchronize();

	auto neighbour_list = readMatrixGPUMemoryCSR(filename);

	auto tempCSC = readMatrixGPUMemoryCSC(filename);
	debugCSC <<<1,1>>> (tempCSC);
	hipDeviceSynchronize();

	double * Si;
	hipMallocManaged(&Si, sizeof(double) * tempCSRCPU->rows);

	double * Si_host = (double *) malloc(sizeof(double) * tempCSRCPU->rows);

	comptueSiHost(tempCSRCPU, tempCSCCPU, Si_host);

	int number_of_blocks = (tempCSRCPU->rows + 1024 - 1) / 1024;
	int number_of_threads = 1024;
	comptueRowColumnAbsSum <<<number_of_blocks, number_of_threads>>> (tempCSR, tempCSC, NULL);
	hipDeviceSynchronize();

	comptueSi<<<number_of_blocks, number_of_threads>>> (tempCSR, tempCSC, Si);
	debugmuij<<<1,1>>> (tempCSR, Si);
	hipDeviceSynchronize();

	sortNeighbourList<<<number_of_blocks, number_of_threads>>> (neighbour_list, tempCSR, Si);

	for(int i = 0; i < tempCSRCPU->rows; i++) {
		for(int j = 0; j < tempCSRCPU->rows; j++) {
			printf("%d %d %lf\n", i, j, muij(i, j, tempCSRCPU, Si_host));
		}
	}
	
	return 0;
}