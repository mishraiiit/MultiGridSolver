#include "hip/hip_runtime.h"
#include "bits/stdc++.h"
#include "Matrix.cu"

using namespace std;

__global__ void debugCOO(MatrixCOO * matrix) {
	for(int i = 0; i < matrix->nnz; i++) {
		printf("%d %d %lf\n", matrix->i[i], matrix->j[i], matrix->val[i]);
	}
}

__global__ void debugCSR(MatrixCSR * matrix) {
	for(int i = 0; i < matrix->rows; i++) {
		for(int j = 0; j < matrix->cols; j++) {
			printf("%lf ", getElementMatrixCSR(matrix, i, j));
		}
		printf("\n");
	}
}

__global__ void debugCSC(MatrixCSC * matrix) {
	for(int i = 0; i < matrix->rows; i++) {
		for(int j = 0; j < matrix->cols; j++) {
			printf("%lf ", getElementMatrixCSC(matrix, i, j));
		}
		printf("\n");
	}
}

__global__ void debugmuij(MatrixCSR * matrix, double * Si) {
	for(int i = 0; i < matrix->rows; i++) {
		for(int j = 0; j < matrix->cols; j++) {
			printf("%d %d %lf\n", i, j, muij(i, j, matrix, Si));
		}
	}
}

int main() {



	TicToc readtime("Read time total");
	readtime.tic();

	string filename = "../../matrices/poisson10000.mtx";

	auto tempCSRCPU = readMatrixCPUMemoryCSR(filename);
	auto tempCSCCPU = readMatrixCPUMemoryCSC(filename);
	auto tempCSR = readMatrixGPUMemoryCSR(filename);
	auto neighbour_list = readMatrixGPUMemoryCSR(filename);
	auto tempCSC = readMatrixGPUMemoryCSC(filename);
	// debugCSR <<<1,1>>> (tempCSR);
	// hipDeviceSynchronize();

	
	// debugCSC <<<1,1>>> (tempCSC);
	// hipDeviceSynchronize();

	readtime.toc();


	TicToc cudaalloctime("cudaalloctime");
	cudaalloctime.tic();

	double * Si;
	hipMallocManaged(&Si, sizeof(double) * tempCSRCPU->rows);

	double * Si_host = (double *) malloc(sizeof(double) * tempCSRCPU->rows);

	cudaalloctime.toc();

	// comptueSiHost(tempCSRCPU, tempCSCCPU, Si_host);

	TicToc rowcolsum("Row Col abs sum");
	rowcolsum.tic();

	int number_of_blocks = (tempCSRCPU->rows + 1024 - 1) / 1024;
	int number_of_threads = 1024;
	comptueRowColumnAbsSum <<<number_of_blocks, number_of_threads>>> (tempCSR, tempCSC, Si);
	hipDeviceSynchronize();

	rowcolsum.toc();


	TicToc sicomputation("Si computation");
	sicomputation.tic();

	comptueSi<<<number_of_blocks, number_of_threads>>> (tempCSR, tempCSC, Si);
	// debugmuij<<<1,1>>> (tempCSR, Si);
	hipDeviceSynchronize();

	sicomputation.toc();

	TicToc sortcomputation("Sort computation");
	sortcomputation.tic();

	sortNeighbourList<<<number_of_blocks, number_of_threads>>> (tempCSR, neighbour_list, Si);
	//printNeighbourList<<<1,1>>> (tempCSR, neighbour_list, Si);

	hipDeviceSynchronize();

	sortcomputation.toc();

	// for(int i = 0; i < tempCSRCPU->rows; i++) {
	// 	for(int j = 0; j < tempCSRCPU->rows; j++) {
	// 		printf("%d %d %lf\n", i, j, muij(i, j, tempCSRCPU, Si_host));
	// 	}
	// }
	
	return 0;
}