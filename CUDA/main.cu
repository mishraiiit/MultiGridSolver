#include "hip/hip_runtime.h"
#include "MatrixIO.cu"
#include "MatrixAccess.cu"
#include "MatrixOperations.cu"
#include "TicToc.cpp"
#include "GPUDebug.cu"
#include "Aggregation.cu"
#include "BFS.cu"
#include "PrefixSum.cu"
#include <hipsparse.h>
#include <string>

// #define SKIP_LEVELS 2

int main(int argc, char * argv[]) {

    std::string matrixname;
    double ktg;
    int npass;
    double tou;
  
    if(argc != 5) {
        printf("Invalid arguments.\n");
        printf("First argument should be matrix file in .mtx format.\n");
        printf("Second argument should be the parameter ktg, default value is 8.\n");
        printf("Third argument should be the parameter npass, default value is 2.\n");
        printf("Fourth argument should be the parameter tou, default value is 4.\n");
        exit(1);
    }

    matrixname = argv[1];
    ktg = std::stod(argv[2]);
    npass = std::stoi(argv[3]);
    tou = std::stod(argv[4]);

    hipsparseHandle_t  cudasparse_handle;
    hipsparseCreate(&cudasparse_handle);


    TicToc readtime("Read time total");
    readtime.tic();

    MatrixCSR * P_cumm = NULL; // output will be in this.
    std::string filename = "../matrices/" + matrixname + ".mtx";

    auto A_CSRCPU = readMatrixCPUMemoryCSR(filename);
    auto A_CSR = deepCopyMatrixCSRCPUtoGPU(A_CSRCPU);

    readtime.toc();

    int nnz_initial = A_CSRCPU->nnz;

    TicToc cudaalloctime("cudaalloctime");
    cudaalloctime.tic();

    float * Si;
    assert(hipMalloc(&Si, sizeof(float) * A_CSRCPU->rows) == hipSuccess);

    bool * ising0;
    assert(hipMalloc(&ising0, sizeof(bool) * A_CSRCPU->rows) == hipSuccess);

    bool * allowed;
    assert(hipMalloc(&allowed, sizeof(bool) * A_CSRCPU->nnz) == hipSuccess);

    int * paired_with;
    assert(hipMalloc(&paired_with, sizeof(int) * A_CSRCPU->rows) == hipSuccess);

    int * useful_pairs;
    assert(hipMalloc(&useful_pairs, sizeof(int) * A_CSRCPU->rows) == hipSuccess);
    
    int * aggregations;
    assert(hipMalloc(&aggregations, sizeof(int) * A_CSRCPU->rows) == hipSuccess);

    int * aggregation_count;
    assert(hipMalloc(&aggregation_count, sizeof(int) * A_CSRCPU->rows) == hipSuccess);

    cudaalloctime.toc();

    TicToc main_timer("Main timer");
    main_timer.tic();

    for(int pass = 1; pass <= npass; pass++) {

        A_CSRCPU = shallowCopyMatrixCSRGPUtoCPU(A_CSR);

        int nnz_now = A_CSRCPU->nnz;
        if(nnz_now <= nnz_initial / tou) break;

        auto neighbour_list = deepCopyMatrixCSRGPUtoGPU(A_CSR);
        auto A_CSC = convertCSRGPU_cudaSparse(A_CSR, cudasparse_handle);
        
        TicToc rowcolsum("Row Col abs sum");
        rowcolsum.tic();
        int number_of_blocks = (A_CSRCPU->rows + 1024 - 1) / 1024;
        int number_of_threads = 1024;
        computeRowColAbsSum <<<number_of_blocks, number_of_threads>>>
        (A_CSR, A_CSC, ising0, ktg, pass);
        hipDeviceSynchronize();
        rowcolsum.toc();

        TicToc sicomputation("Si computation");
        sicomputation.tic();
        comptueSi<<<number_of_blocks, number_of_threads>>> (A_CSR, A_CSC, Si);
        // debugmuij<<<1,1>>> (A_CSR, Si);
        hipDeviceSynchronize();
        sicomputation.toc();

        TicToc bfstime("BFS time...");
        bfstime.tic();
        int max_distance;
        int * bfs_distance = bfs(A_CSRCPU->rows, A_CSR, &max_distance);
        hipDeviceSynchronize();
        bfstime.toc();

        TicToc sortcomputation("Sort computation");
        sortcomputation.tic();
        sortNeighbourList<<<number_of_blocks, number_of_threads>>>
        (A_CSR, neighbour_list, Si, allowed, ktg, ising0);
        hipDeviceSynchronize();
        sortcomputation.toc();

        TicToc aggregationtime("Aggregation time");
        aggregationtime.tic();

        aggregation_initial<<<number_of_blocks, number_of_threads>>>
        (A_CSRCPU->rows, paired_with);
        hipDeviceSynchronize();

        #ifdef SKIP_LEVELS
            int skip_levels = SKIP_LEVELS;
        #else
            int skip_levels = max_distance + 1;
        #endif 

        for(int i = 0; i < skip_levels; i++) {
            aggregation<<<number_of_blocks, number_of_threads>>>
            (A_CSRCPU->rows, neighbour_list, paired_with, allowed, A_CSR, Si, i,
             ising0, bfs_distance, skip_levels);
            hipDeviceSynchronize();
        }

        aggregationtime.toc();

        TicToc get_usefule_pairs_time("Get useful_pairs time");
        get_usefule_pairs_time.tic();
        get_useful_pairs<<<number_of_blocks, number_of_threads>>>
        (A_CSRCPU->rows, paired_with, useful_pairs);
        hipDeviceSynchronize();
        get_usefule_pairs_time.toc();


        TicToc prefix_sum("Sum kernel");
        prefix_sum.tic();
        gpu_prefix_sum(A_CSRCPU->rows, useful_pairs);
        // hipMemcpy(useful_pairs_cpu_prefix, useful_pairs,
        //     sizeof(int) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        prefix_sum.toc();

        TicToc P_matrix_creation_time("Time to P matrix");
        P_matrix_creation_time.tic();

        int nc;
        assert(hipMemcpy(&nc, useful_pairs + A_CSRCPU->rows - 1,
            sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);

        mark_aggregations <<<number_of_blocks, number_of_threads>>>
        (A_CSRCPU->rows, aggregations, useful_pairs);
        hipDeviceSynchronize();

        get_aggregations_count <<< (nc + 1024 - 1) / 1024, 1024 >>>
        (nc, aggregations, paired_with, aggregation_count);
        hipDeviceSynchronize();

        gpu_prefix_sum(nc, aggregation_count);
        int nnz_in_p_matrix;
        assert(hipMemcpy(&nnz_in_p_matrix, aggregation_count + nc - 1,
            sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);
        

        MatrixCSR * P_transpose_shallow_cpu = 
            (MatrixCSR *) malloc(sizeof(MatrixCSR));

        P_transpose_shallow_cpu->rows = nc;
        P_transpose_shallow_cpu->cols = A_CSRCPU->rows;
        P_transpose_shallow_cpu->nnz = nnz_in_p_matrix;

        assert(hipMalloc(&P_transpose_shallow_cpu->i,
            sizeof(int) * (P_transpose_shallow_cpu->rows + 1)) == hipSuccess);
        assert(hipMalloc(&P_transpose_shallow_cpu->j,
            sizeof(int) * (P_transpose_shallow_cpu->nnz)) == hipSuccess);
        assert(hipMalloc(&P_transpose_shallow_cpu->val,
            sizeof(float) * (P_transpose_shallow_cpu->nnz)) == hipSuccess);

        create_p_matrix_transpose <<< (nc + 1024 - 1) / 1024, 1024>>>
        (nc, aggregations, paired_with, aggregation_count,
            P_transpose_shallow_cpu->i, P_transpose_shallow_cpu->j,
            P_transpose_shallow_cpu->val);
        hipDeviceSynchronize();

        MatrixCSR * P_transpose_gpu;
        assert(hipMalloc(&P_transpose_gpu, sizeof(MatrixCSR)) == hipSuccess);
        assert(hipMemcpy(P_transpose_gpu, P_transpose_shallow_cpu, sizeof(MatrixCSR),
            hipMemcpyHostToDevice) == hipSuccess);
        hipDeviceSynchronize();
        P_matrix_creation_time.toc();

        TicToc time_transpose("Time taken csr2csc");
        time_transpose.tic();

        MatrixCSR * P_gpu = transposeCSRGPU_cudaSparse(P_transpose_gpu, cudasparse_handle);
        
        MatrixCSR * newA_gpu = spmatrixmult_cudaSparse(P_transpose_gpu,
            spmatrixmult_cudaSparse(A_CSR, P_gpu, cudasparse_handle),
            cudasparse_handle);

        if(P_cumm == NULL)
            P_cumm = deepCopyMatrixCSRGPUtoGPU(P_gpu);
        else
            P_cumm = spmatrixmult_cudaSparse(P_cumm, P_gpu, cudasparse_handle);

        
        hipFree(bfs_distance);
        freeMatrixCSRGPU(P_gpu);
        freeMatrixCSRGPU(P_transpose_gpu);
        freeMatrixCSRGPU(A_CSR);
        freeMatrixCSCGPU(A_CSC);

        A_CSR = newA_gpu;
    }

    main_timer.toc();

    assert(hipFree(Si) == hipSuccess);
    assert(hipFree(ising0) == hipSuccess);
    assert(hipFree(allowed) == hipSuccess);
    assert(hipFree(paired_with) == hipSuccess);
    assert(hipFree(useful_pairs) == hipSuccess);
    assert(hipFree(aggregations) == hipSuccess);
    assert(hipFree(aggregation_count) == hipSuccess);


    writeMatrixCSRCPU(std::string("../matrices/") + matrixname + \
        std::string("promatrix.mtx"), deepCopyMatrixCSRGPUtoCPU(P_cumm));
 
    freeMatrixCSRGPU(A_CSR);

    return 0;
}
