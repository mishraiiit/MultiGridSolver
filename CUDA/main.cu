#include "hip/hip_runtime.h"
#include "MatrixIO.cu"
#include "MatrixAccess.cu"
#include "MatrixOperations.cu"
#include "TicToc.cpp"
#include "GPUDebug.cu"
#include "Aggregation.cu"
#include "BFS.cu"
#include "PrefixSum.cu"
#include <hipsparse.h>
#include <string>


#define BLELLOCH
// #define THRUST_SORT
// #define DEBUG
// #define SKIP_LEVELS 2

int main(int argc, char * argv[]) {

    std::string matrixname;
    double ktg;
    int npass;
    double tou;
  
    if(argc != 5) {
        printf("Invalid arguments.\n");
        printf("First argument should be matrix file in .mtx format.\n");
        printf("Second argument should be the parameter ktg, default value is 8.\n");
        printf("Third argument should be the parameter npass, default value is 2.\n");
        printf("Fourth argument should be the parameter tou, default value is 4.\n");
        exit(1);
    }

    matrixname = argv[1];
    ktg = std::stod(argv[2]);
    npass = std::stoi(argv[3]);
    tou = std::stod(argv[4]);

    hipsparseHandle_t  cudasparse_handle;
    hipsparseCreate(&cudasparse_handle);


    TicToc readtime("Read time total");
    readtime.tic();

    MatrixCSR * P_cumm = NULL; // output will be in this.
    std::string filename = "../matrices/" + matrixname + ".mtx";

    auto A_CSRCPU = readMatrixCPUMemoryCSR(filename);
    auto A_CSR = deepCopyMatrixCSRCPUtoGPU(A_CSRCPU);

    readtime.toc();

    int nnz_initial = A_CSRCPU->nnz;

    TicToc cudaalloctime("cudaalloctime");
    cudaalloctime.tic();

    float * Si;
    assert(hipMalloc(&Si, sizeof(float) * A_CSRCPU->rows) == hipSuccess);

    int * ising0;
    assert(hipMalloc(&ising0, sizeof(int) * A_CSRCPU->rows) == hipSuccess);

    int * allowed;
    assert(hipMalloc(&allowed, sizeof(int) * A_CSRCPU->nnz) == hipSuccess);

    int * paired_with;
    assert(hipMalloc(&paired_with, sizeof(int) * A_CSRCPU->rows) == hipSuccess);

    int * useful_pairs;
    assert(hipMalloc(&useful_pairs, sizeof(int) * A_CSRCPU->rows) == hipSuccess);
    
    int * aggregations;
    assert(hipMalloc(&aggregations, sizeof(int) * A_CSRCPU->rows) == hipSuccess);

    int * aggregation_count;
    assert(hipMalloc(&aggregation_count, sizeof(int) * A_CSRCPU->rows) == hipSuccess);

    cudaalloctime.toc();

    TicToc main_timer("Main timer");
    main_timer.tic();

    for(int pass = 1; pass <= npass; pass++) {

        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("CSR CPU\n");
            printCSRCPU(deepCopyMatrixCSRGPUtoCPU(A_CSR));
        #endif

        A_CSRCPU = shallowCopyMatrixCSRGPUtoCPU(A_CSR);

        int nnz_now = A_CSRCPU->nnz;
        if(nnz_now <= nnz_initial / tou) break;

        auto neighbour_list = deepCopyMatrixCSRGPUtoGPU(A_CSR);
        auto A_CSC = convertCSRGPU_cudaSparse(A_CSR, cudasparse_handle);
        
        TicToc rowcolsum("Row Col abs sum");
        rowcolsum.tic();
        int number_of_blocks = (A_CSRCPU->rows + 1024 - 1) / 1024;
        int number_of_threads = 1024;
        computeRowColAbsSum <<<number_of_blocks, number_of_threads>>>
        (A_CSR, A_CSC, ising0, ktg, pass);
        hipDeviceSynchronize();
        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Is in G0\n");
            int * temp_g0 = (int * ) malloc(sizeof(int) * A_CSRCPU->rows);
            hipMemcpy(temp_g0, ising0, sizeof(int) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
            for(int i = 0; i < A_CSRCPU->rows; i++) {
                printf("%d ", temp_g0[i]);
            }
            printf("\n");
            free(temp_g0);
        #endif
        rowcolsum.toc();

        TicToc sicomputation("Si computation");
        sicomputation.tic();
        comptueSi<<<number_of_blocks, number_of_threads>>> (A_CSR, A_CSC, Si);
        // debugmuij<<<1,1>>> (A_CSR, Si);
        hipDeviceSynchronize();
        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Si computation\n");
            float * temp_si = (float * ) malloc(sizeof(float) * A_CSRCPU->rows);
            hipMemcpy(temp_si, Si, sizeof(float) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
            for(int i = 0; i < A_CSRCPU->rows; i++) {
                printf("%f ", temp_si[i]);
            }
            printf("\n");
            free(temp_si);
        #endif
        sicomputation.toc();

        TicToc bfstime("BFS time...");
        bfstime.tic();
        int max_distance;
        int * bfs_distance = bfs_work_efficient(A_CSRCPU->rows, A_CSR, &max_distance);
        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("BFS\n");
            int * temp_bfs = (int * ) malloc(sizeof(int) * A_CSRCPU->rows);
            hipMemcpy(temp_bfs, bfs_distance, sizeof(int) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
            for(int i = 0; i < A_CSRCPU->rows; i++) {
                printf("%d ", temp_bfs[i]);
            }
            printf("\n");
            free(temp_bfs);
        #endif
        hipDeviceSynchronize();
        bfstime.toc();

        TicToc sortcomputation("Sort computation");
        sortcomputation.tic();
        sortNeighbourList<<<number_of_blocks, number_of_threads>>>
        (A_CSR, neighbour_list, Si, allowed, ktg, ising0);
        hipDeviceSynchronize();
        #ifdef DEBUG
            printf("\n");
            printf("PASS %d\n", pass);
            printf("neighbour_list\n");
            printCSRCPU(deepCopyMatrixCSRGPUtoCPU(neighbour_list));
        #endif
        sortcomputation.toc();

        TicToc aggregationtime("Aggregation time");
        aggregationtime.tic();

        initialize_array(A_CSRCPU->rows,  paired_with, -1);

        #ifdef SKIP_LEVELS
            int skip_levels = SKIP_LEVELS;
        #else
            int skip_levels = max_distance + 1;
        #endif 

        for(int i = 0; i < skip_levels; i++) {
            aggregation<<<number_of_blocks, number_of_threads>>>
            (A_CSRCPU->rows, neighbour_list, paired_with, allowed, A_CSR, Si, i,
             ising0, bfs_distance, skip_levels);
            hipDeviceSynchronize();
        }

        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Paired with\n");
            int * temp_paired = (int * ) malloc(sizeof(int) * A_CSRCPU->rows);
            hipMemcpy(temp_paired, paired_with, sizeof(int) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
            for(int i = 0; i < A_CSRCPU->rows; i++) {
                printf("%d ", temp_paired[i]);
            }
            printf("\n");
            free(temp_paired);
        #endif

        aggregationtime.toc();

        TicToc get_usefule_pairs_time("Get useful_pairs time");
        get_usefule_pairs_time.tic();
        get_useful_pairs<<<number_of_blocks, number_of_threads>>>
        (A_CSRCPU->rows, paired_with, useful_pairs);
        hipDeviceSynchronize();
        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Useful pairs\n");
            int * temp_useful= (int * ) malloc(sizeof(int) * A_CSRCPU->rows);
            hipMemcpy(temp_useful, useful_pairs, sizeof(int) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
            for(int i = 0; i < A_CSRCPU->rows; i++) {
                printf("%d ", temp_useful[i]);
            }
            printf("\n");
            free(temp_useful);
        #endif
        get_usefule_pairs_time.toc();


        TicToc prefix_sum("Sum kernel");
        prefix_sum.tic();
        prefixSumGPU(useful_pairs, A_CSRCPU->rows);
        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Prefix sum useful pairs\n");
            int * temp_sum = (int * ) malloc(sizeof(int) * A_CSRCPU->rows);
            hipMemcpy(temp_sum, useful_pairs, sizeof(int) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
            for(int i = 0; i < A_CSRCPU->rows; i++) {
                printf("%d ", temp_sum[i]);
            }
            printf("\n");
            free(temp_sum);
        #endif
        // hipMemcpy(useful_pairs_cpu_prefix, useful_pairs,
        //     sizeof(int) * A_CSRCPU->rows, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        prefix_sum.toc();

        TicToc P_matrix_creation_time("Time to P matrix");
        P_matrix_creation_time.tic();

        int nc;
        assert(hipMemcpy(&nc, useful_pairs + A_CSRCPU->rows - 1,
            sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);

        mark_aggregations <<<number_of_blocks, number_of_threads>>>
        (A_CSRCPU->rows, aggregations, useful_pairs);
        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Aggregations\n");
            int * temp_aggregations = (int * ) malloc(sizeof(int) * nc);
            hipMemcpy(temp_aggregations, aggregations, sizeof(int) * nc, hipMemcpyDeviceToHost);
            for(int i = 0; i < nc; i++) {
                printf("%d ", temp_aggregations[i]);
            }
            printf("\n");
            free(temp_aggregations);
        #endif
        hipDeviceSynchronize();

        get_aggregations_count <<< (nc + 1024 - 1) / 1024, 1024 >>>
        (nc, aggregations, paired_with, aggregation_count);
        hipDeviceSynchronize();
        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Aggregations count\n");
            int * temp_aggregations_count = (int * ) malloc(sizeof(int) * nc);
            hipMemcpy(temp_aggregations_count, aggregation_count, sizeof(int) * nc, hipMemcpyDeviceToHost);
            for(int i = 0; i < nc; i++) {
                printf("%d ", temp_aggregations_count[i]);
            }
            printf("\n");
            free(temp_aggregations_count);
        #endif

        prefixSumGPU(aggregation_count, nc);

        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("Aggregations count sum\n");
            int * temp_aggregations_sum = (int * ) malloc(sizeof(int) * nc);
            hipMemcpy(temp_aggregations_sum, aggregation_count, sizeof(int) * nc, hipMemcpyDeviceToHost);
            for(int i = 0; i < nc; i++) {
                printf("%d ", temp_aggregations_sum[i]);
            }
            printf("\n");
            free(temp_aggregations_sum);
        #endif

        int nnz_in_p_matrix;
        assert(hipMemcpy(&nnz_in_p_matrix, aggregation_count + nc - 1,
            sizeof(int), hipMemcpyDeviceToHost) == hipSuccess);
        

        MatrixCSR * P_transpose_shallow_cpu = 
            (MatrixCSR *) malloc(sizeof(MatrixCSR));

        P_transpose_shallow_cpu->rows = nc;
        P_transpose_shallow_cpu->cols = A_CSRCPU->rows;
        P_transpose_shallow_cpu->nnz = nnz_in_p_matrix;

        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("P transpose shallow CPU\n");
            printf("rows: %d, cols: %d, nnz %d\n", P_transpose_shallow_cpu->rows, P_transpose_shallow_cpu->cols, P_transpose_shallow_cpu->nnz);
        #endif

        assert(hipMalloc(&P_transpose_shallow_cpu->i,
            sizeof(int) * (P_transpose_shallow_cpu->rows + 1)) == hipSuccess);
        assert(hipMalloc(&P_transpose_shallow_cpu->j,
            sizeof(int) * (P_transpose_shallow_cpu->nnz)) == hipSuccess);
        assert(hipMalloc(&P_transpose_shallow_cpu->val,
            sizeof(float) * (P_transpose_shallow_cpu->nnz)) == hipSuccess);

        assign<<<1,1>>> (&P_transpose_shallow_cpu->i[0], 0);
        create_p_matrix_transpose <<< (nc + 1024 - 1) / 1024, 1024>>>
        (nc, aggregations, paired_with, aggregation_count,
            P_transpose_shallow_cpu->i, P_transpose_shallow_cpu->j,
            P_transpose_shallow_cpu->val);
        hipDeviceSynchronize();

        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("P transpose row ptr\n");
            int sz = P_transpose_shallow_cpu->rows + 1;
            int * row_ptr = (int * ) malloc(sizeof(int) * sz);
            hipMemcpy(row_ptr, P_transpose_shallow_cpu->i, sizeof(int) * sz, hipMemcpyDeviceToHost);
            for(int i = 0; i < sz; i++) {
                printf("%d ", row_ptr[i]);
            }
            printf("\n");
            free(row_ptr);
        #endif

         #ifdef DEBUG
            {
                printf("PASS %d\n", pass);
                printf("P transpose j\n");
                int sz = P_transpose_shallow_cpu->nnz;
                int * temp = (int * ) malloc(sizeof(int) * sz);
                hipMemcpy(temp, P_transpose_shallow_cpu->j, sizeof(int) * sz, hipMemcpyDeviceToHost);
                for(int i = 0; i < sz; i++) {
                    printf("%d ", temp[i]);
                }
                printf("\n");
                free(temp);
            }
        #endif

        #ifdef DEBUG
            {
                printf("PASS %d\n", pass);
                printf("P transpose val\n");
                int sz = P_transpose_shallow_cpu->nnz;
                float * temp = (float * ) malloc(sizeof(float) * sz);
                hipMemcpy(temp, P_transpose_shallow_cpu->val, sizeof(float) * sz, hipMemcpyDeviceToHost);
                for(int i = 0; i < sz; i++) {
                    printf("%f ", temp[i]);
                }
                printf("\n");
                free(temp);
            }
        #endif

        MatrixCSR * P_transpose_gpu;
        assert(hipMalloc(&P_transpose_gpu, sizeof(MatrixCSR)) == hipSuccess);
        assert(hipMemcpy(P_transpose_gpu, P_transpose_shallow_cpu, sizeof(MatrixCSR),
            hipMemcpyHostToDevice) == hipSuccess);

        #ifdef DEBUG
            printf("PASS %d\n", pass);
            printf("P_transpose\n");
            printCSRCPU(deepCopyMatrixCSRGPUtoCPU(P_transpose_gpu));
        #endif

        hipDeviceSynchronize();
        P_matrix_creation_time.toc();

        TicToc time_transpose("Time taken csr2csc");
        time_transpose.tic();

        MatrixCSR * P_gpu = transposeCSRGPU_cudaSparse(P_transpose_gpu, cudasparse_handle);
        
        MatrixCSR * newA_gpu = spmatrixmult_cudaSparse(P_transpose_gpu,
            spmatrixmult_cudaSparse(A_CSR, P_gpu, cudasparse_handle),
            cudasparse_handle);

        if(P_cumm == NULL)
            P_cumm = deepCopyMatrixCSRGPUtoGPU(P_gpu);
        else
            P_cumm = spmatrixmult_cudaSparse(P_cumm, P_gpu, cudasparse_handle);

        
        hipFree(bfs_distance);
        freeMatrixCSRGPU(P_gpu);
        freeMatrixCSRGPU(P_transpose_gpu);
        freeMatrixCSRGPU(A_CSR);
        freeMatrixCSCGPU(A_CSC);

        A_CSR = newA_gpu;
    }

    main_timer.toc();

    assert(hipFree(Si) == hipSuccess);
    assert(hipFree(ising0) == hipSuccess);
    assert(hipFree(allowed) == hipSuccess);
    assert(hipFree(paired_with) == hipSuccess);
    assert(hipFree(useful_pairs) == hipSuccess);
    assert(hipFree(aggregations) == hipSuccess);
    assert(hipFree(aggregation_count) == hipSuccess);


    writeMatrixCSRCPU(std::string("../matrices/") + matrixname + \
        std::string("promatrix.mtx"), deepCopyMatrixCSRGPUtoCPU(P_cumm));
 
    freeMatrixCSRGPU(A_CSR);

    return 0;
}
