#include "hip/hip_runtime.h"
#ifndef PREFIX_SUM_GPU
#define PREFIX_SUM_GPU
#include "scan.cu"

__global__ void copyToInput(int * out, int * in, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= n) return;
	if(i == n - 1) {
		in[i] += out[i];
	} else {
		in[i] = out[i + 1];
	}
}

void prefixSumGPU(int * in, int n) {
	int * out;
	hipMalloc(&out, sizeof(int) * n);
	sum_scan_blelloch(out, in, n);
	copyToInput <<< (n + 1024 - 1) / 1024, 1024 >>> (out, in, n);
	hipFree(out);
}


// void prefixSumGPUExclusive(int * in, int n) {
// 	int * out;
// 	hipMalloc(&out, sizeof(int) * n);
// 	sum_scan_blelloch(out, in, n);
// 	hipMemcpy(in, out, sizeof(int) * n, hipMemcpyDeviceToDevice);
// 	hipFree(out);
// }


void prefixSumCPU(int * in, int n) {
	int * cpu_in = (int *) malloc(n * sizeof(int));
	hipMemcpy(cpu_in, in, sizeof(int) * n, hipMemcpyDeviceToHost);
	int sum = 0;
	for(int i = 0; i < n; i++) {
		cpu_in[i] += sum;
		sum = cpu_in[i];
	}
	hipMemcpy(in, cpu_in, sizeof(int) * n, hipMemcpyHostToDevice);
}

#endif